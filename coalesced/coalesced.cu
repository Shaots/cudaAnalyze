#include "hip/hip_runtime.h"
#include <iostream>
#include "coalesced.cuh"

__global__ void noCoalescedAccess(float* odata, float* idata, int size) {
    int len = (size - 1) / blockDim.x + 1;
    for (int i = 0; i < len; ++i) {
        odata[threadIdx.x * len + i] = idata[threadIdx.x * len + i];
    }
}

__global__ void coalescedAccess(float* odata, float* idata, int size) {
    int len = (size - 1) / blockDim.x + 1;
    for (int i = 0; i < len; ++i) {
        odata[i * blockDim.x + threadIdx.x] = idata[i * blockDim.x + threadIdx.x];
    }
}

void test(coalesced f, int warm, int times){
    size_t len = 100000;
    float* arr = new float[len];
    for (size_t i = 0; i < len; ++i) {
        arr[i] = 5.5f;
    }
    float* d_arr_in;
    hipMalloc((void**)&d_arr_in, len * sizeof(float));
    hipMemcpy(d_arr_in, arr, len, hipMemcpyHostToDevice);

    float* d_arr_out;
    hipMalloc((void**)&d_arr_out, len * sizeof(float));

    dim3 threadDim = 32;

    hipEvent_t start, stop;
    float       time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int i = 0; i < warm; ++i) {
        f<<<1, threadDim>>>(d_arr_out, d_arr_in, len);
    }

    hipDeviceSynchronize();

    hipEventRecord(start, 0);

    for (int i = 0; i < times; ++i) {
        f<<<1, threadDim>>>(d_arr_out, d_arr_in, len);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    float* out = new float[len];
    hipMemcpy(out, d_arr_out, len, hipMemcpyDeviceToHost);

    std::cout << time / times << " ms" << std::endl;

    hipFree(d_arr_in);
    hipFree(d_arr_out);
    delete[] arr;
    delete[] out;
}