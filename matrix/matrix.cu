#include "hip/hip_runtime.h"
#include "matrix.cuh"

__global__ void simpleMultiply(const int* a, const int* b, int* c, int M, int K, int N) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int sum = 0;
    if (col >= N || row >= M) return;
    for (int i = 0; i < K; ++i) {
        sum += a[row * K + i] * b[i * N + col];
    }
    c[row * N + col] = sum;
}

__global__ void coalescedMultiply(const int* a, const int* b, int* c, int M, int K, int N) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Here is no condition return !!!

    __shared__ int sh[32][32];
    int            sum = 0;
    int            numSubMatrix = (K >> 5) + 1;
    for (int sub = 0; sub < numSubMatrix; ++sub) {
        // We cannot write
        // if (row >= M || col >= N || threadIdx.x + sub * 32 >= K)
        if (row >= M || threadIdx.x + sub * 32 >= K)
            sh[threadIdx.y][threadIdx.x] = 0;
        else
            sh[threadIdx.y][threadIdx.x] = a[row * K + sub * 32 + threadIdx.x];
        __syncwarp();
        for (int i = 0; i < 32; ++i) {
            sum += sh[threadIdx.y][i] * b[(sub * 32 + i) * N + col];
        }
        __syncwarp();
    }
    if (row < M && col < N) c[row * N + col] = sum;
}


__global__ void sharedMultiply(const int* a, const int* b, int* c, int M, int K, int N) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Here is no condition return !!!

    __shared__ int shA[32][32];
    __shared__ int shB[32][32];
    int            sum = 0;
    int            numSubMatrix = (K >> 5) + 1;
    for (int sub = 0; sub < numSubMatrix; ++sub) {
        // We cannot write
        // if (row >= M || col >= N || threadIdx.x + sub * 32 >= K)
        if (row >= M || sub * 32 + threadIdx.x >= K)
            shA[threadIdx.y][threadIdx.x] = 0;
        else
            shA[threadIdx.y][threadIdx.x] = a[row * K + sub * 32 + threadIdx.x];

        if (col >= N || sub * 32 + threadIdx.y >= K)
            shB[threadIdx.y][threadIdx.x] = 0;
        else
            shB[threadIdx.y][threadIdx.x] = b[(sub * 32 + threadIdx.y) * N + col];
        __syncthreads();
        for (int i = 0; i < 32; ++i) {
            sum += shA[threadIdx.y][i] * shB[i][threadIdx.x];
        }
        __syncthreads();
    }
    if (row < M && col < N) c[row * N + col] = sum;
}

void testMultiply(multiFunc func, int warm, int times) {
    std::srand(std::time({})); // use current time as seed for random generator

    int M = 1000;
    int K = 2000;
    int N = 3000;

    int* a = new int[M * K];
    int* b = new int[K * N];
    int* c = new int[M * N];
    int* cpuMultiply = new int[M * N];

    for (int i = 0; i < M * K; ++i) a[i] = std::rand() % 20;
    for (int i = 0; i < K * N; ++i) b[i] = std::rand() % 20;

    multiply(a, b, cpuMultiply, M, K, N);

    int* d_a;
    int* d_b;
    int* d_c;
    hipMalloc((void**)&d_a, M * K * sizeof(int));
    hipMalloc((void**)&d_b, K * N * sizeof(int));
    hipMalloc((void**)&d_c, M * N * sizeof(int));

    hipMemcpy(d_a, a, M * K * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, K * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim = {32, 32};
    dim3 gridDim = {(N - 1) / blockDim.x + 1, (M - 1) / blockDim.y + 1};

    hipEvent_t start, stop;
    float       time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int i = 0; i < warm; ++i) {
        func<<<gridDim, blockDim>>>(d_a, d_b, d_c, M, K, N);
    }

    hipDeviceSynchronize();

    hipEventRecord(start, 0);

    for (int i = 0; i < times; ++i) {
        func<<<gridDim, blockDim>>>(d_a, d_b, d_c, M, K, N);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(c, d_c, M * N * sizeof(int), hipMemcpyDeviceToHost);

    bool flag = true;
    for (int i = 0; i < M * N; ++i) {
        if (c[i] != cpuMultiply[i]) {
            std::cout << "Error: " << i << " " << c[i] << " " << cpuMultiply[i];
            flag = false;
            break;
        }
    }
    std::cout << std::boolalpha << flag << std::endl;

    std::cout << time / times << " ms" << std::endl;

    delete[] a;
    delete[] b;
    delete[] c;
    delete[] cpuMultiply;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

void multiply(const int* a, const int* b, int* c, int M, int K, int N) {
    int sum;
    for(int row = 0; row < M; ++row) {
        for (int col = 0; col < N; ++col) {
            sum = 0;
            for (int k = 0; k < K; ++k) {
                sum += a[row * K + k] * b[k * N + col];
            }
            c[row * N + col] = sum;
        }
    }
}